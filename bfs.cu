#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <hip/hip_runtime.h>
#define N 10
#define BLOCK_SIZE 1024

using namespace std;

__global__ void bfs(int endNode, int* neighbour_array, int* offset, int* visited, int* q, int* prev)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
      if(i == 1024){
        i = 0;
      }
      if (q[i] != -1) {
          int currentNode = q[i];
              for(int y = offset[i]; y < offset[i+1]; y++){
                  int neighbour = neighbour_array[y];
                  if (visited[neighbour] == 0) {
                      visited[neighbour] = 1;
                      q[neighbour] = neighbour;
                      prev[neighbour] = currentNode;
                  }
              }
      }
}


void edgeStreamToCSR(std::vector<int>& neighbour, std::vector<int>& offset)
{
    std::ifstream inputFile("output.txt");
    vector<vector<int>> adjlist;

    if(!inputFile) {
        std::cerr <<"Unable to open file for reading\n";
    }
    int numVertices, numEdges;
    inputFile >> numVertices >> numEdges;

    int u, v;

    for(int i = 0; i < numVertices; ++i) {
       adjlist.push_back(std::vector<int>());
    }

    for(int i = 0; i < numEdges; ++i) {
        inputFile >> u >> v;
        adjlist[u].push_back(v);
        adjlist[v].push_back(u);
    }
    offset.resize(numVertices + 1);

    offset[0] = 0;

    for(int i = 0; i < adjlist.size(); ++i) {
        offset[i+1] = offset[i] + adjlist[i].size();
        for(int j = 0; j < adjlist[i].size(); ++j) {
            neighbour.push_back(adjlist[i][j]);
        }
    }
}

void createInputGraph(int width, int depth, int& numVertices, int& numEdges, std::vector<int>& neighbour, vector<int>& offset)
{
    std::ofstream outFile("output.txt");
    if (!outFile) {
        std::cerr << "Error: Unable to create file. \n";
    }

    numEdges = width*depth;
    numVertices = numEdges + 1;

    outFile << numVertices <<" " << numEdges << std::endl;
    for(int i = 1; i <= width; ++i)
        outFile << "0" <<" " << i << std::endl;
    depth--;

    int start_index;
    int itr_num = 0;

    while(depth--) {
        start_index = width * itr_num;

        for(int i = 1; i <= width; ++i) {
            outFile << start_index + i << " " << start_index + i + width << std::endl;
            //std::cout << start_index + i << " " << start_index + i + width << std::endl;
        }
        itr_num++;
    }

    outFile.close();


    edgeStreamToCSR(neighbour, offset);

}

/*void bfs(vector<vector<int> >& adjList, int startNode, vector<bool>& visited)
{
    queue<int> q;
    visited[startNode] = true;
    q.push(startNode);
    while (!q.empty()) {
        int currentNode = q.front();
        q.pop();
        cout << currentNode << " ";
        for (int neighbor : adjList[currentNode]) {
            if (!visited[neighbor]) {
                visited[neighbor] = true;
                q.push(neighbor);
            }
        }
    }
}*/

int main() {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;

    int width = 33;
    int depth = 33;

    int startNode = 0;
    int endNode = width*depth;

    std::vector<int> neighbour;
    std::vector<int> offset;
    std::vector<int> q;
    hipError_t status = hipSuccess;

    int threadsPerBlock;
    int blocksPerGrid;

    threadsPerBlock = BLOCK_SIZE;

    blocksPerGrid = 1;

    int numVertices, numEdges;
    createInputGraph(width, depth, numVertices, numEdges, neighbour, offset);

    int *visited = (int *)malloc(numVertices * sizeof(int));
    int *prev = (int *)malloc(numVertices * sizeof(int));

    for (int i=0; i<numVertices; i++)
    {
      visited[i] = 0;
      prev[i] = -1;
    }
    for (int i=0; i<numVertices; i++)
    {
      q.push_back(-1);
    }
    q[0] = 0;
    visited[0] = 1;
    prev[0] = 0;

    int *neighbourD = NULL;
    int *offsetD = NULL;
    int *visitedD = NULL;
    int *qD = NULL;
    int *prevD = NULL;
    std::size_t neighbour_size = sizeof(int) * neighbour.size();
    std::size_t offset_size = sizeof(int) * offset.size();
    std::size_t visited_size = sizeof(int) * numVertices;
    std::size_t q_size = sizeof(int) * q.size();
    std::size_t prev_size = sizeof(int) * numVertices;

    status = hipMalloc((void **)&neighbourD, neighbour_size);
    status = hipMalloc((void **)&offsetD, offset_size);
    status = hipMalloc((void **)&visitedD, visited_size);
    status = hipMalloc((void **)&qD, q_size);
    status = hipMalloc((void **)&prevD, prev_size);

    hipMemcpy(neighbourD, neighbour.data(), neighbour_size, hipMemcpyHostToDevice);
    hipMemcpy(offsetD, offset.data(), offset_size, hipMemcpyHostToDevice);
    hipMemcpy(visitedD, visited, visited_size, hipMemcpyHostToDevice);
    hipMemcpy(qD, q.data(), q_size, hipMemcpyHostToDevice);
    hipMemcpy(prevD, prev, prev_size, hipMemcpyHostToDevice);

    hipEventRecord(start,0);

    while(visited[endNode] != 1){
        bfs<<<blocksPerGrid, threadsPerBlock>>>(endNode, neighbourD, offsetD, visitedD, qD, prevD);
        hipMemcpy(visited, visitedD, visited_size, hipMemcpyDeviceToHost);
    }

    hipEventRecord(stop,0);
    hipEventSynchronize (stop);
    hipEventElapsedTime (&time, start, stop);

    hipEventDestroy (start);
    hipEventDestroy (stop);
    cout<<"Time taken : "<<time<<" milli seconds"<<endl;

    //cudaMemcpy(visited, visitedD, visited_size, cudaMemcpyDeviceToHost);
    hipMemcpy(prev, prevD, prev_size, hipMemcpyDeviceToHost);

    int current = prev[endNode];
    printf("%d --> ", endNode);
    for(int i = 0; i<prev_size; i++){
        if(current != startNode){
            printf("%d --> ", current);
            current = prev[current];
        }
    }
    printf("%d ", startNode);

    return EXIT_SUCCESS;
}
